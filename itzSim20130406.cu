#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cudabyexample/common/book.h"

//#define NUMNEURNS	10000
//#define TIME 1000

//const int neuronsPerBlock
//const int blocksPerGrid = imin( 32, (NUMNEURNS+neuronsPerBlock-1) / neuronsPerBlock );

//#define INPUT 0

//itzSim20130406


__global__ void test(float *I) {
	int neuronIndex;
	neuronIndex = threadIdx.x + blockIdx.x * blockDim.x; 
	
	I[neuronIndex] = 0;
}


__global__ void updateNeuronState(int *spikeNeuron, int *nrnPreIndexArray, int *nrnPostIndexArray, int *postSynapseList, int *axonState, int *axonDelay, float *synapseState, float *I, float *v, float *u, float *a, float *d, float *dt, float *time) {

/*
int *spikeNeuron, 1xnNeurons 1s or 0s if a spike fired or not (can do a bool?)
int *nrnPostIndexArray,	1xnNeurons+1 start indicies for the post synaptic neuron list
int *nrnPreIndexArray,	1xnNeurons+1 start indicies for the pre synaptic neuron list
int *postSynapseList, 1xnConnections lists the post synaptic neuron targets for every neuron, indexed by nrnIndexArray
int *axonState, 1xnConnections
int *axonDelay, 1xnConnections
float *synapseState, 1xnConnections
float *I, 1xnNeurons
float *v, 1xnNeruons
float *u, 1xnNeurons
float *a, 1xnNeurons
float *d, 1xnNeurons
float *dt, 1x1
float *time 1x1
*/

	int neuronIndex;
	
	neuronIndex = threadIdx.x + blockIdx.x * blockDim.x; 

	// add incoming currents
	float synapticInput = (50 - v[neuronIndex]) * I[neuronIndex];

	// update synaptic input
	for (int iConnection = nrnPreIndexArray[neuronIndex]; iConnection < nrnPreIndexArray[neuronIndex+1]; iConnection ++){
        if (synapseState[iConnection] > 0){
            synapticInput = synapticInput + synapseState[iConnection] * (50 - v[neuronIndex]);
			}
        else if (synapseState[iConnection] < 0){
            synapticInput = synapticInput - synapseState[iConnection] * (-75 - v[neuronIndex]);
			}
    }
    
	//synapticInput = 0.0;
	//v[neuronIndex] = synapticInput;


	v[neuronIndex] = v[neuronIndex] + 0.5 * dt[0] * ( (0.04 * v[neuronIndex] + 5 ) * v[neuronIndex] + 140 - u[neuronIndex] + synapticInput );    // for
	v[neuronIndex] = v[neuronIndex] + 0.5 * dt[0] * ( (0.04 * v[neuronIndex] + 5 ) * v[neuronIndex] + 140 - u[neuronIndex] + synapticInput );    // stability time
	u[neuronIndex] = u[neuronIndex] + a[neuronIndex] * (0.2 * v[neuronIndex]-u[neuronIndex]);                   // step is 0.5 ms
	      
	
	
	spikeNeuron[neuronIndex] = 0;
	if (v[neuronIndex] > 40.0){
		v[neuronIndex] = -65.0;
		u[neuronIndex] = u[neuronIndex] + d[neuronIndex];
		spikeNeuron[neuronIndex] = 1;
		for (int iConnection = nrnPostIndexArray[neuronIndex]; iConnection < nrnPostIndexArray[neuronIndex+1]; iConnection ++){
			axonState[postSynapseList[iConnection]] = axonDelay[postSynapseList[iConnection]];
		}

	
	}

	
		
}



__global__ void updateSynapticState(int *connectionsPerThread, int *nConnections, float *synapseState, float *tau, int *axonState, float *synapseWeight) {

/*
int   *axonState 1xnConnections
float *synapseState 1xnConnections
float *synapseWeight 1xnConnections
float *tau 1xnConnections
*/

	int connectionSet = threadIdx.x + blockIdx.x * blockDim.x; 

    
    //connectionsPerThread = ceil(nConnections/(gridDim.x * blockDim.x));
    
	int iConnection;
	
    for (iConnection = 
	(connectionSet * connectionsPerThread[0]);
	 iConnection < ((connectionSet+1) * connectionsPerThread[0]);
	  iConnection ++){
        
        if (iConnection < nConnections[0]){
            
            if (synapseState[iConnection] != 0) {
                synapseState[iConnection] = synapseState[iConnection]-synapseState[iConnection]*tau[iConnection];
				}
            else if (synapseState[iConnection] > -0.0001 && synapseState[iConnection] < -0.0001)
			{
                synapseState = 0;
            }
            
            if (axonState[iConnection] > 1){
                axonState[iConnection] = axonState[iConnection]-1;
				}
            else if (axonState[iConnection] == 1){
                axonState[iConnection] = axonState[iConnection]-1;
                synapseState[iConnection] = synapseState[iConnection] + synapseWeight[iConnection];
				}
            else if (axonState[iConnection] < 0){
                axonState[iConnection] = 0;
            }

        }

    }

}


